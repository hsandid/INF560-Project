
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
//#include <cuda_runtime.h>

// includes
//#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
//#include <helper_cuda.h>       // helper functions for CUDA error checking and initialization

#define MAX(X,Y) ((X) > (Y) ? (X) : (Y))  /* utility function */

__global__ void vecAddThreeKernel(char *a, char *b, char *c, char *res, int N)
{
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i<N)
    {
        res[i] = a[i]+b[i] + c[i];
    }
}

__global__ void kernComputeForces(double*cuda_xpos_array, double*cuda_ypos_array, double*cuda_mass_array, double*cuda_xforce,double*cuda_yforce,int size)
{
    int i;
    i = blockIdx.x * blockDim.x + threadIdx.x;
    //int j;
    //j = blockDim.y * blockIdx.y + threadIdx.y;
    
    if (i<size)
    {
    int j;
    for (j=0;j<size;j++)
    {
        double x_sep, y_sep, dist_sq, grav_base;

        printf("LOOP on INDEX %d PASS %d\n",i,j);

        x_sep = cuda_xpos_array[j] - cuda_xpos_array[i];
        y_sep =  cuda_ypos_array[j] - cuda_ypos_array[i];
        dist_sq = MAX((x_sep * x_sep) + (y_sep * y_sep), 0.01);

        /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
        grav_base =  0.01 * (cuda_mass_array[i]) * (cuda_mass_array[j]) / dist_sq;

        cuda_xforce[i] += grav_base * x_sep;
        cuda_yforce[i] += grav_base * y_sep;
    }

    //printf("X block id %d block dim %d threadID %d index %d val %f %f %f %f %f\n",blockIdx.x ,blockDim.x, threadIdx.x, i, cuda_xpos_array[i], cuda_ypos_array[i],cuda_mass_array[i], cuda_xforce[i], cuda_yforce[i]);
    // if (i<size)
    // {
    //     cuda_xforce[i] += cuda_xpos_array[i]+cuda_ypos_array[i] + cuda_mass_array[i];
    //     cuda_yforce[i] += cuda_xpos_array[i]+cuda_ypos_array[i] + cuda_mass_array[i];
    // }
    }
    
}

// Take a step back and compartementalize
// 1: Take pointer reference to array of particles, number of particles to treat
// start index and end index
// 2: Allocate arrays to store double of xpos, ypos, mass, xforce, yforce  on CPU side
// 3: Allocate 5 arrays on CUDA side to store double of xpos, ypos, mass; and xforce and yforce
// 4: OP

//5: Copy xforce and yforce results back into CPU memory
//6: Iterate to update particles forces as needed
//7: Free all arrays on NVIDIA side
//8: Return

// int startIndex, int endIndex, particle_t *particleArray
extern "C" void GPUComputeForce()
{
    // Save x coordinates int
    double *xpos_array   =(double *) malloc(sizeof(double)*(5));
    double * ypos_array = (double *)malloc(sizeof(double)*(5));
    double * mass_array = (double *)malloc(sizeof(double)*(5));
    double *xforce =  (double *)malloc(sizeof(double)*(5));
    double *yforce = (double *) malloc(sizeof(double)*(5));

    int i;
    for (i=0;i<5;i++)
    {
        xpos_array[i] = i;
        ypos_array[i] = i;
        mass_array[i] = i;
        xforce[i] = 0.0;
        yforce[i] = 0.0;
    }

    // Save x coordinates int
    double *cuda_xpos_array  ;
    double * cuda_ypos_array;
    double * cuda_mass_array;
    double *cuda_xforce ;
    double *cuda_yforce ;

    

    hipMalloc((void **)&cuda_xpos_array, 5*sizeof(double));
    hipMalloc((void **)&cuda_ypos_array, 5*sizeof(double));
    hipMalloc((void **)&cuda_mass_array,5*sizeof(double));
    hipMalloc((void **)&cuda_xforce, 5*sizeof(double));
    hipMalloc((void **)&cuda_yforce, 5*sizeof(double));

    hipMemcpy(cuda_xpos_array, xpos_array, 5*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cuda_ypos_array, ypos_array, 5*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cuda_mass_array, mass_array, 5*sizeof(double), hipMemcpyHostToDevice);

    kernComputeForces<<<2 , 5 >>>(cuda_xpos_array, cuda_ypos_array, cuda_mass_array, cuda_xforce, cuda_yforce, 5);

    hipMemcpy(xforce, cuda_xforce, 5*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(yforce, cuda_yforce, 5*sizeof(double), hipMemcpyDeviceToHost);

    //Create loop to pass into particles array

    // test loop
    // for (i=0;i<50;i++)
    // {
    //     printf("%f %f \n",xforce[i], yforce[i]);
    // }
    // Save x coordinates int
    hipFree(cuda_xpos_array);
    hipFree( cuda_ypos_array);
    hipFree( cuda_mass_array);
    hipFree(cuda_xforce);
    hipFree(cuda_yforce);
}

/* Function computing the final string to print */
void compute_string( char * res, char * a, char * b, char *c, int length ) 
{





char * d_a ;
char * d_b ;
char * d_c ;
char * d_res;

hipMalloc((void **)&d_a, length*sizeof(char));
hipMalloc((void **)&d_b, length*sizeof(char));
hipMalloc((void **)&d_c, length*sizeof(char));
hipMalloc((void **)&d_res, length*sizeof(char));

hipMemcpy(d_a, a, length*sizeof(char), hipMemcpyHostToDevice);
hipMemcpy(d_b, b, length*sizeof(char), hipMemcpyHostToDevice);
hipMemcpy(d_c, c, length*sizeof(char), hipMemcpyHostToDevice);
//cudaMemcpy(d_res, res, length*sizeof(char), cudaMemcpyHostToDevice);

// Launch kernel
vecAddThreeKernel<<< 4, 20 >>>(d_a, d_b, d_c, d_res, length);
//vecAddThreeKernel(d_a, d_b, d_c, d_res, length);

hipMemcpy(res, d_res, length*sizeof(char), hipMemcpyDeviceToHost);

hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
hipFree(d_res);

}
